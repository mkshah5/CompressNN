#include "cuSZp_timer.h"

TimingGPU::TimingGPU() { privateTimingGPU = new PrivateTimingGPU;  }

TimingGPU::~TimingGPU() { }

void TimingGPU::StartCounter()
{
    hipEventCreate(&((*privateTimingGPU).start));
    hipEventCreate(&((*privateTimingGPU).stop));
    hipEventRecord((*privateTimingGPU).start,0);
}

void TimingGPU::StartCounterFlags()
{
    int eventflags = hipEventBlockingSync;

    hipEventCreateWithFlags(&((*privateTimingGPU).start),eventflags);
    hipEventCreateWithFlags(&((*privateTimingGPU).stop),eventflags);
    hipEventRecord((*privateTimingGPU).start,0);
}

// Gets the counter in ms
float TimingGPU::GetCounter()
{
    float time;
    hipEventRecord((*privateTimingGPU).stop, 0);
    hipEventSynchronize((*privateTimingGPU).stop);
    hipEventElapsedTime(&time,(*privateTimingGPU).start,(*privateTimingGPU).stop);
    return time;
}
