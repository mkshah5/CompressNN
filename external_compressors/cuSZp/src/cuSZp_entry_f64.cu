#include "cuSZp_entry_f64.h"
#include "cuSZp_f64.h"

void SZp_compress_hostptr_f64(double* oriData, unsigned char* cmpBytes, size_t nbEle, size_t* cmpSize, double errorBound)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f64;
    int gsize = (nbEle + bsize * cmp_chunk_f64 - 1) / (bsize * cmp_chunk_f64);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f64;

    // Initializing global memory for GPU compression.
    double* d_oriData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_oriData, sizeof(double)*pad_nbEle);
    hipMemcpy(d_oriData, oriData, sizeof(double)*pad_nbEle, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpData, sizeof(double)*pad_nbEle);
    hipMallocManaged((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(double));

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_compress_kernel_f64<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU.  
    *cmpSize = (size_t)d_cmpOffset[cmpOffSize-1] + (nbEle+31)/32;
    hipMemcpy(cmpBytes, d_cmpData, *cmpSize*sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Free memory that is used.
    hipFree(d_oriData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}


void SZp_decompress_hostptr_f64(double* decData, unsigned char* cmpBytes, size_t nbEle, size_t cmpSize, double errorBound)
{
    // Data blocking.
    int bsize = dec_tblock_size_f64;
    int gsize = (nbEle + bsize * dec_chunk_f64 - 1) / (bsize * dec_chunk_f64);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * dec_chunk_f64;

    // Initializing global memory for GPU compression.
    double* d_decData;
    unsigned char* d_cmpData;
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_decData, sizeof(double)*pad_nbEle);
    hipMemset(d_decData, 0, sizeof(double)*pad_nbEle);
    hipMalloc((void**)&d_cmpData, sizeof(double)*pad_nbEle);
    hipMemcpy(d_cmpData, cmpBytes, sizeof(unsigned char)*cmpSize, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // Initializing CUDA Stream.
    hipStream_t stream;
    hipStreamCreate(&stream);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_decompress_kernel_f64<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpData, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Move data back to CPU.
    hipMemcpy(decData, d_decData, sizeof(double)*nbEle, hipMemcpyDeviceToHost);

    // Free memoy that is used.
    hipFree(d_decData);
    hipFree(d_cmpData);
    hipFree(d_cmpOffset);
    hipFree(d_flag);
    hipStreamDestroy(stream);
}


void SZp_compress_deviceptr_f64(double* d_oriData, unsigned char* d_cmpBytes, size_t nbEle, size_t* cmpSize, double errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = cmp_tblock_size_f64;
    int gsize = (nbEle + bsize * cmp_chunk_f64 - 1) / (bsize * cmp_chunk_f64);
    int cmpOffSize = gsize + 1;
    int pad_nbEle = gsize * bsize * cmp_chunk_f64;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMallocManaged((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);
    hipMemset(d_oriData + nbEle, 0, (pad_nbEle - nbEle) * sizeof(double));

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_compress_kernel_f64<<<gridSize, blockSize, 0, stream>>>(d_oriData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Obtain compression ratio and move data back to CPU.  
    *cmpSize = (size_t)d_cmpOffset[cmpOffSize-1] + (nbEle+31)/32;

    // Free memory that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);
}


void SZp_decompress_deviceptr_f64(double* d_decData, unsigned char* d_cmpBytes, size_t nbEle, size_t cmpSize, double errorBound, hipStream_t stream)
{
    // Data blocking.
    int bsize = dec_tblock_size_f64;
    int gsize = (nbEle + bsize * dec_chunk_f64 - 1) / (bsize * dec_chunk_f64);
    int cmpOffSize = gsize + 1;

    // Initializing global memory for GPU compression.
    unsigned int* d_cmpOffset;
    int* d_flag;
    hipMalloc((void**)&d_cmpOffset, sizeof(unsigned int)*cmpOffSize);
    hipMemset(d_cmpOffset, 0, sizeof(unsigned int)*cmpOffSize);
    hipMalloc((void**)&d_flag, sizeof(int)*cmpOffSize);
    hipMemset(d_flag, 0, sizeof(int)*cmpOffSize);

    // cuSZp GPU compression.
    dim3 blockSize(bsize);
    dim3 gridSize(gsize);
    SZp_decompress_kernel_f64<<<gridSize, blockSize, 0, stream>>>(d_decData, d_cmpBytes, d_cmpOffset, d_flag, errorBound, nbEle);
    hipDeviceSynchronize();

    // Free memoy that is used.
    hipFree(d_cmpOffset);
    hipFree(d_flag);
}